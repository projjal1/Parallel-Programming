/*
Problem : Add two matrix in CUDA 
Author : Projjal Gop
*/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define n 512

__global__ void add(int *a,int *b,int *c)
{
	c[blockIdx.x]=a[blockIdx.x]+b[blockIdx.x];
}

int main(void)
{
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	int size=n * sizeof(int);
	 
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);
	
	a=(int *)malloc(size); 
	b=(int *)malloc(size); 
	c=(int *)malloc(size);
	
	for(int i=0;i<n;i++)
	{
		a[i]=rand();
		printf("%d , ",a[i]);
	}
	for(int i=0;i<n;i++)
	{
		b[i]=rand();
		printf("%d , ",b[i]);
	}
		
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	
	add<<<n,1>>>(d_a,d_b,d_c);
	
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	
	free(a); free(b); free(c);
	
	for (int i=0;i<n;i++)
		printf("%d ",c[i]);
	return 0;
}